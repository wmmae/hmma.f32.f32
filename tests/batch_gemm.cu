#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <wmma_extension/hmma_f32_f32.hpp>

namespace {
constexpr unsigned warp_size = 32;

// SMEM_M * SMEM_N must be larger than or equal to BLOCK_SIZE
template <unsigned SMEM_M, unsigned SMEM_N, unsigned BLOCK_SIZE>
__device__ void dmem2smem(
		float* const dst_smem,
		const unsigned m, const unsigned n,
		const float* const src_dmem, const unsigned ld
		) {
	if (m == SMEM_M && n == SMEM_N) {
		if (SMEM_M & 0b11 == 0) {
			for (unsigned i = 0; i < SMEM_M * SMEM_N; i += BLOCK_SIZE * 4) {
				const auto j = i + threadIdx.x * 4;
				const auto j_m = j % SMEM_M;
				const auto j_n = j / SMEM_M;
				const auto mem_index = j_m + j_n * ld;

				const auto tmp_v4 = *reinterpret_cast<const float4*>(&src_dmem[mem_index]);

				*reinterpret_cast<float4*>(&dst_smem[j]) = tmp_v4;
			}
		} else {
			for (unsigned i = 0; i < SMEM_M * SMEM_N; i += BLOCK_SIZE) {
				const auto j = i + threadIdx.x;
				const auto j_m = j % SMEM_M;
				const auto j_n = j / SMEM_M;

				dst_smem[j] = src_dmem[j_m + j_n * ld];
			}
		}
	} else {
		for (unsigned i = 0; i < SMEM_M * SMEM_N; i += BLOCK_SIZE) {
			const auto j = i + threadIdx.x;
			const auto j_m = j % SMEM_M;
			const auto j_n = j / SMEM_M;

			float v = 0.f;
			if (j_m < m && j_n < n) {
				v = src_dmem[j_m + j_n * ld];
			}

			dst_smem[j] = v;
		}
	}
}

// SMEM_M * SMEM_N must be larger than or equal to BLOCK_SIZE
template <unsigned SMEM_M, unsigned SMEM_N, unsigned BLOCK_SIZE>
__device__ void smem2dmem(
		float* const dst_dmem, const unsigned ld,
		const unsigned m, const unsigned n,
		const float* const src_smem,
		const float alpha, const float beta
		) {
	if (beta == 0.f) {
		if (m == SMEM_M && n == SMEM_N) {
			if (SMEM_M & 0b11 == 0) {
				for (unsigned i = 0; i < SMEM_M * SMEM_N; i += BLOCK_SIZE * 4) {
					const auto j = i + threadIdx.x * 4;
					const auto j_m = j % SMEM_M;
					const auto j_n = j / SMEM_M;
					const auto mem_index = j_m + j_n * ld;

					auto tmp_v4 = make_float4(
							src_smem[j + 0],
							src_smem[j + 1],
							src_smem[j + 2],
							src_smem[j + 3]
							);
					tmp_v4.x *= alpha;
					tmp_v4.y *= alpha;
					tmp_v4.z *= alpha;
					tmp_v4.w *= alpha;

					*reinterpret_cast<float4*>(&dst_dmem[mem_index]) = tmp_v4;
				}
			} else {
				for (unsigned i = 0; i < SMEM_M * SMEM_N; i += BLOCK_SIZE) {
					const auto j = i + threadIdx.x;
					const auto j_m = j % SMEM_M;
					const auto j_n = j / SMEM_M;

					dst_dmem[j_m + j_n * ld] = alpha * src_smem[j];
				}
			}
		} else {
			for (unsigned i = 0; i < SMEM_M * SMEM_N; i += BLOCK_SIZE) {
				const auto j = i + threadIdx.x;
				const auto j_m = j % SMEM_M;
				const auto j_n = j / SMEM_M;

				if (j_m < m && j_n < n) {
					dst_dmem[j_m + j_n * ld] = alpha * src_smem[j];
				}
			}
		}
	} else {
		// beta is not zero
		if (m == SMEM_M && n == SMEM_N) {
			for (unsigned i = 0; i < SMEM_M * SMEM_N; i += BLOCK_SIZE) {
				const auto j = i + threadIdx.x;
				const auto j_m = j % SMEM_M;
				const auto j_n = j / SMEM_M;

				const auto dmem_offset = j_m + j_n * ld;
				dst_dmem[dmem_offset] = alpha * src_smem[j] + beta * dst_dmem[dmem_offset];
			}
		} else {
			for (unsigned i = 0; i < SMEM_M * SMEM_N; i += BLOCK_SIZE) {
				const auto j = i + threadIdx.x;
				const auto j_m = j % SMEM_M;
				const auto j_n = j / SMEM_M;

				if (j_m < m && j_n < n) {
					const auto dmem_offset = j_m + j_n * ld;
					dst_dmem[dmem_offset] = alpha * src_smem[j] + beta * dst_dmem[dmem_offset];
				}
			}
		}
	}
}

// SMEM_M * SMEM_N must be larger than or equal to BLOCK_SIZE
template <unsigned SMEM_M, unsigned SMEM_N, unsigned BLOCK_SIZE>
__device__ void fill_zero(
		float* const dst_smem
		) {
	for (unsigned i = 0; i < SMEM_M * SMEM_N; i += BLOCK_SIZE) {
		const auto j = i + threadIdx.x;
		dst_smem[j] = 0.f;
	}
}

template <
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned WARP_M,
	unsigned WARP_N,
	unsigned WARP_K,
	unsigned BLOCK_SIZE,
	class FRAGMENT_T,
	class TC_Policy>
__device__ void mma_core(
		float* const c_smem,
		float* const a_smem,
		float* const b_smem
		) {
#pragma unroll
	for (unsigned w = 0; w < (SMEM_M * SMEM_N / (WARP_M * WARP_N)); w += BLOCK_SIZE / warp_size) {
		const auto wi = w + threadIdx.x / warp_size;
		const auto wi_m = (wi % (SMEM_M / WARP_M)) * WARP_M;
		const auto wi_n = (wi / (SMEM_M / WARP_M)) * WARP_N;

		// Load A
		mtk::wmma::mma_f32::fragment<nvcuda::wmma::matrix_a, WARP_M, WARP_N, WARP_K, FRAGMENT_T, nvcuda::wmma::row_major, TC_Policy> frag_a;
		const auto a_smem_offset = wi_m * SMEM_K + 0;
		mtk::wmma::mma_f32::load_matrix_sync(frag_a, a_smem + a_smem_offset, SMEM_K, false);

		// Load B
		mtk::wmma::mma_f32::fragment<nvcuda::wmma::matrix_b, WARP_M, WARP_N, WARP_K, FRAGMENT_T, nvcuda::wmma::col_major, TC_Policy> frag_b;
		const auto b_smem_offset = wi_n * SMEM_K + 0;
		mtk::wmma::mma_f32::load_matrix_sync(frag_b, b_smem + b_smem_offset, SMEM_K, false);

		// Load C
		mtk::wmma::mma_f32::fragment<nvcuda::wmma::accumulator, WARP_M, WARP_N, WARP_K, FRAGMENT_T, void, TC_Policy> frag_c;
		const auto c_smem_offset = wi_m + wi_n * SMEM_M;
		mtk::wmma::mma_f32::load_matrix_sync(frag_c, c_smem + c_smem_offset, SMEM_M, nvcuda::wmma::mem_col_major);

#pragma unroll
		for (unsigned wi_k = WARP_K; wi_k < SMEM_K; wi_k += WARP_K) {
			// mma
			mtk::wmma::mma_f32::mma_sync(frag_c, frag_a, frag_b, frag_c);

			// Load A
			mtk::wmma::mma_f32::fragment<nvcuda::wmma::matrix_a, WARP_M, WARP_N, WARP_K, FRAGMENT_T, nvcuda::wmma::row_major, TC_Policy> frag_a;
			const auto a_smem_offset = wi_m * SMEM_K + wi_k;
			mtk::wmma::mma_f32::load_matrix_sync(frag_a, a_smem + a_smem_offset, SMEM_K, false);

			// Load B
			mtk::wmma::mma_f32::fragment<nvcuda::wmma::matrix_b, WARP_M, WARP_N, WARP_K, FRAGMENT_T, nvcuda::wmma::col_major, TC_Policy> frag_b;
			const auto b_smem_offset = wi_n * SMEM_K + wi_k;
			mtk::wmma::mma_f32::load_matrix_sync(frag_b, b_smem + b_smem_offset, SMEM_K, false);
		}
		// mma
		mtk::wmma::mma_f32::mma_sync(frag_c, frag_a, frag_b, frag_c);
		mtk::wmma::mma_f32::store_matrix_sync(c_smem + c_smem_offset, frag_c, SMEM_M, nvcuda::wmma::mem_col_major, false);
	}
}

// This kernel function computes batched matrix-matrix multiplication
// A needs to be row major, and B needst to be col major
template <
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned WARP_M,
	unsigned WARP_N,
	unsigned WARP_K,
	unsigned BLOCK_SIZE,
	class FRAGMENT_T,
	class TC_Policy>
__global__ void bgemm_kernel(
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const float alpha,
		const float* const* const a_ptr, const unsigned lda,
		const float* const* const b_ptr, const unsigned ldb,
		const float beta,
		float* const* const c_ptr, const unsigned ldc
		) {
	constexpr unsigned num_stages = 2;
	// Sharedm memory
	extern __shared__ float smem[];
	float* const a_smem = smem;
	float* const b_smem = a_smem + SMEM_M * SMEM_K * num_stages;
	float* const c_smem = b_smem + SMEM_K * SMEM_N * num_stages;
	float* const a_smem_array[num_stages] = {a_smem, a_smem + SMEM_M * SMEM_K};
	float* const b_smem_array[num_stages] = {b_smem, b_smem + SMEM_K * SMEM_N};

	for (unsigned bn = 0; bn < n; bn += SMEM_N) {
		for (unsigned bm = 0; bm < m; bm += SMEM_M) {
			// Load A from device memory to shared memory
			const auto real_bm = min(SMEM_M, m - bm);
			const auto real_bk = min(SMEM_K, k);
			const auto a_dmem_offset = bm * lda;
			const float* const a_dmem = a_ptr[blockIdx.x];
			// Load row major A using a loader for col major
			dmem2smem<SMEM_K, SMEM_M, BLOCK_SIZE>(a_smem_array[0], real_bk, real_bm, a_dmem + a_dmem_offset, lda);

			// Load B from global memory to shared memory
			const auto real_bn = min(SMEM_N, n - bn);
			const auto b_dmem_offset = bn * ldb;
			const float* const b_dmem = b_ptr[blockIdx.x];
			dmem2smem<SMEM_K, SMEM_N, BLOCK_SIZE>(b_smem_array[0], real_bk, real_bn, b_dmem + b_dmem_offset, ldb);

			//fill_zero<SMEM_M, SMEM_N, BLOCK_SIZE>(c_smem);
			unsigned stage = 0;
#pragma unroll
			for (unsigned bk = SMEM_K; bk < k; bk += SMEM_K) {
				// MMA
				__syncthreads();
				mma_core<SMEM_M, SMEM_N, SMEM_K, WARP_M, WARP_N, WARP_K, BLOCK_SIZE, FRAGMENT_T, TC_Policy>(c_smem, a_smem_array[stage], b_smem_array[stage]);

				stage = 1 - stage;

				// Load A from device memory to shared memory
				const auto real_bk = min(SMEM_K, k - bk);
				const auto a_dmem_offset = bm * lda + bk;
				dmem2smem<SMEM_K, SMEM_M, BLOCK_SIZE>(a_smem_array[stage], real_bk, real_bm, a_dmem + a_dmem_offset, lda);

				// Load B from global memory to shared memory
				const auto b_dmem_offset = bn * ldb + bk;
				dmem2smem<SMEM_K, SMEM_N, BLOCK_SIZE>(b_smem_array[stage], real_bk, real_bn, b_dmem + b_dmem_offset, ldb);
			} // loop bk

			// MMA
			__syncthreads();
			mma_core<SMEM_M, SMEM_N, SMEM_K, WARP_M, WARP_N, WARP_K, BLOCK_SIZE, FRAGMENT_T, TC_Policy>(c_smem, a_smem_array[stage], b_smem_array[stage]);

			__syncthreads();
			const auto c_dmem_offset = bm + bn * ldc;
			float* const c_dmem = c_ptr[blockIdx.x];
			smem2dmem<SMEM_M, SMEM_N, BLOCK_SIZE>(c_dmem + c_dmem_offset, ldc, real_bm, real_bn, c_smem, alpha, beta);
		} // loop bn
	} // loop bm
}

template <
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned WARP_M,
	unsigned WARP_N,
	unsigned WARP_K,
	unsigned BLOCK_SIZE,
	class FRAGMENT_T,
	class TC_Policy>
void bgemm(
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const float alpha,
		const float* const* const a_ptr, const unsigned lda,
		const float* const* const b_ptr, const unsigned ldb,
		const float beta,
		float* const* const c_ptr, const unsigned ldc,
		const unsigned batch_size
		) {
	// Set shared memory size
	const auto shared_memory_size = ((SMEM_M * SMEM_K + SMEM_K * SMEM_N) * 2 + SMEM_M * SMEM_N) * sizeof(float);
	hipFuncSetAttribute(reinterpret_cast<const void*>(&(bgemm_kernel<SMEM_M), SMEM_N, SMEM_K, WARP_M, WARP_N, WARP_K, BLOCK_SIZE, FRAGMENT_T, TC_Policy>), hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory_size);

	// Launch
	bgemm_kernel<SMEM_M, SMEM_N, SMEM_K, WARP_M, WARP_N, WARP_K, BLOCK_SIZE, FRAGMENT_T, TC_Policy><<<batch_size, BLOCK_SIZE, shared_memory_size>>>(
			m, n, k,
			alpha,
			a_ptr, lda,
			b_ptr, ldb,
			beta,
			c_ptr, ldc
			);
}

template <
	unsigned SMEM_M,
	unsigned SMEM_N,
	unsigned SMEM_K,
	unsigned WARP_M,
	unsigned WARP_N,
	unsigned WARP_K,
	unsigned BLOCK_SIZE>
void test_batched_sgemm(
		const unsigned m,
		const unsigned n,
		const unsigned k,
		const unsigned batch_size
		) {
	std::printf("!-- %s\n", __func__);
	using FRAGMENT_T = half;
	using TC_Policy = mtk::wmma::mma_f32::detail::default_policy<FRAGMENT_T, mtk::wmma::mma_f32::op_with_error_correction, mtk::wmma::mma_f32::op_mma>::type;

	float **d_a_ptr_array;
	float **d_b_ptr_array;
	float **d_c_ptr_array;
	hipMalloc(&d_a_ptr_array, sizeof(float*) * batch_size);
	hipMalloc(&d_b_ptr_array, sizeof(float*) * batch_size);
	hipMalloc(&d_c_ptr_array, sizeof(float*) * batch_size);

	float **h_a_ptr_array;
	float **h_b_ptr_array;
	float **h_c_ptr_array;
	hipHostMalloc(&h_a_ptr_array, sizeof(float*) * batch_size);
	hipHostMalloc(&h_b_ptr_array, sizeof(float*) * batch_size);
	hipHostMalloc(&h_c_ptr_array, sizeof(float*) * batch_size);

	// Host memory for initializing
	float* init_matrix;
	hipHostMalloc(&init_matrix, sizeof(float) * m * n * k / (std::min(m, std::min(n, k))));
	for (unsigned i = 0; i < batch_size; i++) {
		// Allocate device memory and set
		float *d_a_ptr;
		float *d_b_ptr;
		float *d_c_ptr;
		hipMalloc(&d_a_ptr, sizeof(float) * m * k);
		hipMalloc(&d_b_ptr, sizeof(float) * k * n);
		hipMalloc(&d_c_ptr, sizeof(float) * m * n);
		h_a_ptr_array[i] = d_a_ptr;
		h_b_ptr_array[i] = d_b_ptr;
		h_c_ptr_array[i] = d_c_ptr;

		// Initialize matrices
		// A
		for (unsigned j = 0; j < m * k; j++) init_matrix[j] = j / static_cast<float>(m * k);
		hipMemcpy(d_a_ptr, init_matrix, sizeof(float) * m * k, hipMemcpyDefault);
		// B
		for (unsigned j = 0; j < k * n; j++) init_matrix[j] = j / static_cast<float>(k * n);
		hipMemcpy(d_b_ptr, init_matrix, sizeof(float) * k * n, hipMemcpyDefault);
		// C
		for (unsigned j = 0; j < m * n; j++) init_matrix[j] = 0.f;
		hipMemcpy(d_c_ptr, init_matrix, sizeof(float) * m * n, hipMemcpyDefault);
	}
	hipHostFree(init_matrix);

	// Copy the pointer array to the device
	hipMemcpy(d_a_ptr_array, h_a_ptr_array, sizeof(float*) * batch_size, hipMemcpyDefault);
	hipMemcpy(d_b_ptr_array, h_b_ptr_array, sizeof(float*) * batch_size, hipMemcpyDefault);
	hipMemcpy(d_c_ptr_array, h_c_ptr_array, sizeof(float*) * batch_size, hipMemcpyDefault);
	std::printf("Start evaluation\n");

	hipDeviceSynchronize();
	bgemm<SMEM_M, SMEM_N, SMEM_K, WARP_M, WARP_N, WARP_K, BLOCK_SIZE, FRAGMENT_T, TC_Policy>(
			m, n, k,
			1.f,
			d_a_ptr_array, m,
			d_b_ptr_array, n,
			0.f,
			d_c_ptr_array, k,
			batch_size
			);
	hipDeviceSynchronize();


	std::printf("-------\n");

	// evaluate the last batch matrix
	float* last_a_ptr;
	float* last_b_ptr;
	float* last_c_ptr;
	hipHostMalloc(&last_a_ptr, sizeof(float) * m * k);
	hipHostMalloc(&last_b_ptr, sizeof(float) * k * n);
	hipHostMalloc(&last_c_ptr, sizeof(float) * m * n);
	hipMemcpy(last_a_ptr, h_a_ptr_array[batch_size - 1], sizeof(float) * m * k, hipMemcpyDefault);
	hipMemcpy(last_b_ptr, h_b_ptr_array[batch_size - 1], sizeof(float) * k * n, hipMemcpyDefault);
	hipMemcpy(last_c_ptr, h_c_ptr_array[batch_size - 1], sizeof(float) * m * n, hipMemcpyDefault);
	double base_norm = 0.;
	double diff_norm = 0.;
#pragma omp parallel for collapse(2) reduction(+: base_norm) reduction(+: diff_norm)
	for (unsigned i = 0; i < m; i++) {
		for (unsigned j = 0; j < n; j++) {
			double c = 0.;
			for (unsigned l = 0; l < k; l++) {
				c += static_cast<double>(last_a_ptr[l + i * k]) * static_cast<double>(last_b_ptr[l + j * k]);
			}
			const auto diff = last_c_ptr[i + j * m] - c;
			const auto base = c;
			base_norm += base * base;
			diff_norm += diff * diff;
		}
	}
	hipFree(last_a_ptr);
	hipFree(last_b_ptr);
	hipFree(last_c_ptr);

	hipDeviceSynchronize();
	// evaluation of computing performance
	constexpr unsigned test_count = 1lu << 2;
	const auto start_clock = std::chrono::system_clock::now();
	for (unsigned c = 0; c < test_count; c++) {
		bgemm<SMEM_M, SMEM_N, SMEM_K, WARP_M, WARP_N, WARP_K, BLOCK_SIZE, FRAGMENT_T, TC_Policy>(
				m, n, k,
				1.f,
				d_a_ptr_array, m,
				d_b_ptr_array, n,
				0.f,
				d_c_ptr_array, k,
				batch_size
				);
	}
	hipDeviceSynchronize();
	const auto end_clock = std::chrono::system_clock::now();
	const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_clock - start_clock).count() * 1e-6 / test_count;
	const auto complexity = 2lu * static_cast<std::size_t>(m) * static_cast<std::size_t>(n) * static_cast<std::size_t>(k) * static_cast<std::size_t>(batch_size);
	const auto performance = complexity / elapsed_time / (1lu << 40);


	std::printf("-------\n");
	std::printf("%15s: (%u, %u, %u)\n", "Size", m, n, k);
	std::printf("%15s: %u\n", "Batch size", batch_size);
	std::printf("%15s: %lu byte\n", "Shared memory", sizeof(float) * (SMEM_M * SMEM_K + SMEM_K * SMEM_N + SMEM_M * SMEM_N));
	std::printf("%15s: %e s\n", "Time", elapsed_time);
	std::printf("%15s: %e TFlop/s\n", "Performance", performance);
	std::printf("%15s: %e\n", "Error", std::sqrt(diff_norm / base_norm));

	// Free
	for (unsigned i = 0; i < batch_size; i++) {
		hipFree(h_a_ptr_array[i]);
		hipFree(h_b_ptr_array[i]);
		hipFree(h_c_ptr_array[i]);
	}
	hipFree(d_a_ptr_array);
	hipFree(d_b_ptr_array);
	hipFree(d_c_ptr_array);
	hipHostFree(h_a_ptr_array);
	hipHostFree(h_b_ptr_array);
	hipHostFree(h_c_ptr_array);
}
} // noname napespace

int main() {
	test_batched_sgemm<128, 64, 16, 64, 16, 16, 256>(1024, 1024, 1024, 512);
}
