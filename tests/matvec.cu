#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include "utils.hpp"

#ifdef WMMAE_USE_NVCUDA_NAMESPACE
namespace f32_namespace = nvcuda;
#else
namespace f32_namespace = mtk;
#endif

template <class T, class ErrorCorrection>
constexpr double error_threshold = 0.0;
template <>
constexpr double error_threshold<half                         , mtk::wmma::op_with_error_correction   > = 1e-5;
template <>
constexpr double error_threshold<nvcuda::wmma::precision::tf32, mtk::wmma::op_with_error_correction   > = 1e-5;
template <>
constexpr double error_threshold<half                         , mtk::wmma::op_without_error_correction> = 1e-2;
template <>
constexpr double error_threshold<nvcuda::wmma::precision::tf32, mtk::wmma::op_without_error_correction> = 1e-2;

template <unsigned N, class T, class Policy>
__global__ void matvec_kernel(float* const y_ptr, const float* const a_ptr, const float* const x_ptr) {
	__shared__ float smem[N * N];
	mtk::test_utils::fill_zero(smem, N * N);

	f32_namespace::wmma::fragment_f32<nvcuda::wmma::matrix_a   , N, N, N, T, nvcuda::wmma::col_major, Policy> frag_a;
	f32_namespace::wmma::fragment_f32<nvcuda::wmma::matrix_b   , N, N, N, T, nvcuda::wmma::col_major, Policy> frag_x;
	f32_namespace::wmma::fragment_f32<nvcuda::wmma::accumulator, N, N, N, T, void                   , Policy> frag_y;
	// Load A
	mtk::test_utils::copy_matrix(smem, N, a_ptr, N, N, N);
	f32_namespace::wmma::load_matrix_sync(frag_a, smem, N);

	// Load X
	mtk::test_utils::copy_matrix(smem, N, x_ptr, N, N, 1);
	f32_namespace::wmma::fill_zero(frag_x);
	f32_namespace::wmma::load_vector(frag_x, smem);

	// mma
	f32_namespace::wmma::mma_sync(frag_y, frag_a, frag_x);

	// Store D
	f32_namespace::wmma::store_vector(smem, frag_y, nvcuda::wmma::mem_col_major);
	mtk::test_utils::copy_matrix(y_ptr, N, smem, N, N, 1);
}

template <unsigned N, class T, class Policy>
void test_matvec() {
	float *hX, *hY, *hA;
	hipHostMalloc(&hX, N     * sizeof(float));
	hipHostMalloc(&hY, N     * sizeof(float));
	hipHostMalloc(&hA, N * N * sizeof(float));

	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

	for (unsigned i = 0; i < N * N; i++) {
			hA[i] = dist(mt);
	}
	for (unsigned i = 0; i < N; i++) {
			hX[i] = dist(mt);
	}
	hipDeviceSynchronize();

	matvec_kernel<N, T, Policy><<<1, mtk::test_utils::warp_size>>>(hY, hA, hX);

	hipDeviceSynchronize();

	double max_error = 0.;
	for (unsigned n = 0; n < N; n++) {
		double cor_d = 0.;
		for (unsigned k = 0; k < N; k++) {
			cor_d += static_cast<double>(hA[k * N + n]) * static_cast<double>(hX[k]);
		}

		max_error = std::max(max_error, std::abs(cor_d - hY[n]));
	}

	std::printf(
			"[Type:%5s, N:%3u, Policy<%7s,%9s,%2u,%2u,%2u>] max_error: %e (%6s)\n",
			mtk::test_utils::to_string<T>().c_str(),
			N,
			std::is_same<typename Policy::op, mtk::wmma::op_wmma>::value ? "op_wmma" : "op_mma",
			std::is_same<typename Policy::error_correction, mtk::wmma::op_with_error_correction>::value ? "{w/ ec}" : "{w/o ec}",
			Policy::m,
			Policy::n,
			Policy::k,
			max_error,
			(max_error < error_threshold<T, typename Policy::error_correction> ? "PASSED" : "FAILED")
			);

	hipHostFree(hA);
	hipHostFree(hX);
	hipHostFree(hY);
}

int main() {
	// wmma FP16 test
	test_matvec<32, half, typename mtk::wmma::detail::default_policy<half, mtk::wmma::op_with_error_correction   , mtk::wmma::op_wmma>::type>();
	test_matvec<32, half, typename mtk::wmma::detail::default_policy<half, mtk::wmma::op_without_error_correction, mtk::wmma::op_wmma>::type>();

#ifdef TEST_TF32
	// wmma TF32 test
	test_matvec<32, nvcuda::wmma::precision::tf32, typename mtk::wmma::detail::default_policy<nvcuda::wmma::precision::tf32, mtk::wmma::op_with_error_correction   , mtk::wmma::op_wmma>::type>();
	test_matvec<32, nvcuda::wmma::precision::tf32, typename mtk::wmma::detail::default_policy<nvcuda::wmma::precision::tf32, mtk::wmma::op_without_error_correction, mtk::wmma::op_wmma>::type>();
#endif
}
